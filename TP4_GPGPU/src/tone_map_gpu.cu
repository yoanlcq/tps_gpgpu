#include <tone_map.hpp>
#include <rgbhsv.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

#include <ChronoGPU.hpp>
#include <ScopedChrono.hpp>

typedef ScopedChrono<ChronoGPU> ScopedChronoGPU;

void tone_map_gpu_rgb(Rgb24* __restrict__ dst, const Rgb24* __restrict__ src, uint32_t w, uint32_t h) {
}
