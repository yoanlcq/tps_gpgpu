#include "hip/hip_runtime.h"
#include <tone_map.hpp>
#include <rgbhsv.hpp>
#include <handle_cuda_error.hpp>
#include <ChronoGPU.hpp>
#include <ScopedChrono.hpp>

#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

// TODO 
// - GPU: Profiler et sauver les résultats pour le rapport !
// - GPU: Once it works:
//   - Use a texture;
//   - Use constant or shared memory;
//   - Minimize atomicAdd()s;

#define L TONEMAP_LEVELS

__global__ static void rgb_to_hsv_then_put_in_histogram(
          uint32_t* const __restrict__ dev_hist,
          float*    const __restrict__ dev_hue,
          float*    const __restrict__ dev_sat,
          float*    const __restrict__ dev_val,
    const uchar3*   const __restrict__ dev_rgb,
    const uint32_t w, const uint32_t h
) {
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t i = y * w + x;

    if(x >= w || y >= h)
        return;

    const uchar3 rgb = dev_rgb[i];
    const float r = rgb.x / 255.f;
    const float g = rgb.y / 255.f;
    const float b = rgb.z / 255.f;
    const float cmax = fmaxf(fmaxf(r, g), b);
    const float cmin = fminf(fminf(r, g), b);
    const float delta = cmax - cmin;
    const float EPSILON = 0.0001f;

    if(delta <= EPSILON || cmax <= EPSILON) {
        dev_hue[i] = 0.f;
        dev_sat[i] = 0.f;
    } else {
        float hue = 0.f;

             if(r >= cmax) hue = 0 + (g-b) / delta;
        else if(g >= cmax) hue = 2 + (b-r) / delta;
        else               hue = 4 + (r-g) / delta;

        if(hue < 0)
            hue += 6;

        hue *= 60;
        dev_hue[i] = hue;
        dev_sat[i] = delta / cmax;
    }

    const float val = cmax;
    dev_val[i] = val;

    const uint32_t l = val * 255;
    atomicAdd(&dev_hist[l], 1);
}

// CDF = Cumulative Distribution Function
__global__ static void generate_cdf_via_inclusive_scan_histogram(
          uint32_t* const __restrict__ dev_cdf, 
    const uint32_t* const __restrict__ dev_hist
) {
    // Slide 20 of
    // http://people.cs.vt.edu/yongcao/teaching/cs5234/spring2013/slides/Lecture10.pdf

    // Assume L/2 threads, and only 1 block
    __shared__ uint32_t shared_cdf[L];
    shared_cdf[threadIdx.x*2 + 0] = dev_hist[threadIdx.x*2 + 0];
    shared_cdf[threadIdx.x*2 + 1] = dev_hist[threadIdx.x*2 + 1];
    __syncthreads();

    // Reduction step
    for(uint32_t stride=1 ; stride <= L/2 ; stride *= 2) {
        const uint32_t i = (threadIdx.x+1) * stride * 2 - 1;
        if(i < L) {
            shared_cdf[i] += shared_cdf[i - stride];
        }
        __syncthreads();
    }
    // Post scan step
    for(int32_t stride=L/4 ; stride > 0 ; stride /= 2) {
        const uint32_t i = (threadIdx.x+1) * stride * 2 - 1;
        if(i + stride < L) {
            shared_cdf[i + stride] += shared_cdf[i];
        }
        __syncthreads();
    }
    dev_cdf[threadIdx.x*2 + 0] = shared_cdf[threadIdx.x*2 + 0];
    dev_cdf[threadIdx.x*2 + 1] = shared_cdf[threadIdx.x*2 + 1];
}

__global__ static void tone_map_then_hsv_to_rgb(
         uchar3*    const __restrict__ dev_rgb,
    const float*    const __restrict__ dev_hue,
    const float*    const __restrict__ dev_sat,
    const float*    const __restrict__ dev_val, 
    const uint32_t* const __restrict__ dev_cdf,
    const uint32_t w, const uint32_t h
) {
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t i = y * w + x;

    if(x >= w || y >= h)
        return;

    const uint32_t l = dev_val[i] * (L-1);
    const float val = (dev_cdf[l] - dev_cdf[0]) / float(w*h);
    const float hue = dev_hue[i];
    const float sat = dev_sat[i];

    const float hp = hue / 60;
    const float c = val * sat; // chroma
    const float X = c * (1 - fabsf(fmodf(hp, 2) - 1));
    float r, g, b;
    switch((int)hp) {
    case 0: r = c, g = X, b = 0; break;
    case 1: r = X, g = c, b = 0; break;
    case 2: r = 0, g = c, b = X; break;
    case 3: r = 0, g = X, b = c; break;
    case 4: r = X, g = 0, b = c; break;
    case 5: r = c, g = 0, b = X; break;
    default: r = g = b = 0; break;
    }
    const float m = val - c;
    r += m, g += m, b += m;
    dev_rgb[i] = make_uchar3(r * 255, g * 255, b * 255);
}

__global__ static void sanity_check_kernel() {}

typedef ScopedChrono<ChronoGPU> ScopedChronoGPU;

void tone_map_gpu_rgb(Rgb24* __restrict__ host_dst, const Rgb24* __restrict__ host_src, uint32_t w, uint32_t h) {

    // Vérifier que l'architecture du code est compatible avec ce PC. Ca m'avait silencieusement trahi.
    sanity_check_kernel<<<1,1>>>();
    handle_cuda_error(hipGetLastError());

    uchar3*   dev_rgb = NULL;
    float*    dev_hue = NULL;
    float*    dev_sat = NULL;
    float*    dev_val = NULL;
    uint32_t* dev_hist = NULL;
    uint32_t* dev_cdf = NULL;

    {
        char txt[128];
        uint32_t total_bytes = w * h * (3+4+4+4) + L * (4+4);
        snprintf(txt, sizeof txt, "GPU: Allocating %u bytes (~%u MiB)",
            total_bytes, total_bytes / (1024 * 1024)
        );
        ScopedChronoGPU chr(txt);
        handle_cuda_error(hipMalloc(&dev_rgb, w * h * sizeof dev_rgb[0]));
        handle_cuda_error(hipMalloc(&dev_hue, w * h * sizeof dev_hue[0]));
        handle_cuda_error(hipMalloc(&dev_sat, w * h * sizeof dev_sat[0]));
        handle_cuda_error(hipMalloc(&dev_val, w * h * sizeof dev_val[0]));
        handle_cuda_error(hipMalloc(&dev_hist, L * sizeof dev_hist[0]));
        handle_cuda_error(hipMalloc(&dev_cdf, L * sizeof dev_cdf[0]));

        handle_cuda_error(hipMemset(dev_hist, 0, L * sizeof dev_hist[0]));
    }

    {
        ScopedChronoGPU chr("GPU: Uploading RGB data");
        assert(sizeof(host_src[0]) == sizeof(dev_rgb[0]));
        handle_cuda_error(hipMemcpy(dev_rgb, host_src, w * h * sizeof dev_rgb[0], hipMemcpyHostToDevice));
    }

    // 16*16 = 256 threads/tile
    // 32*32 = 1024 threads/tile
    const dim3 img_threads(32, 32);
    const dim3 img_blocks(
        (w + img_threads.x - 1) / img_threads.x,
        (h + img_threads.y - 1) / img_threads.y
    );

#ifdef NDEBUG
#define check_kernel_error()
#else
#define check_kernel_error() handle_cuda_error(hipGetLastError())
#endif

    {
        ScopedChronoGPU chr("GPU: RGB to HSV, then incrementing histogram slots");
        rgb_to_hsv_then_put_in_histogram<<<img_blocks, img_threads>>>(
            dev_hist, dev_hue, dev_sat, dev_val, dev_rgb, w, h
        );
    }
    check_kernel_error();

    {
        ScopedChronoGPU chr("GPU: Generating CDF via inclusive scan of histogram");
        generate_cdf_via_inclusive_scan_histogram<<<1, L/2>>>(
            dev_cdf, dev_hist
        );
    }
    check_kernel_error();

    {
        ScopedChronoGPU chr("GPU: Tone mapping, then HSV to RGB");
        tone_map_then_hsv_to_rgb<<<img_blocks, img_threads>>>(
            dev_rgb, dev_hue, dev_sat, dev_val, dev_cdf, w, h
        );
    }
    check_kernel_error();

    {
        ScopedChronoGPU chr("GPU: Downloading RGB data");
        assert(sizeof(host_dst[0]) == sizeof(dev_rgb[0]));
        handle_cuda_error(hipMemcpy(host_dst, dev_rgb, w * h * sizeof dev_rgb[0], hipMemcpyDeviceToHost));
    }

    {
        ScopedChronoGPU chr("GPU: Freeing memory");
        handle_cuda_error(hipFree(dev_rgb));
        handle_cuda_error(hipFree(dev_hue));
        handle_cuda_error(hipFree(dev_sat));
        handle_cuda_error(hipFree(dev_val));
        handle_cuda_error(hipFree(dev_hist));
        handle_cuda_error(hipFree(dev_cdf));
    }
}

#if 0 // code de test
void tone_map_gpu_rgb(Rgb24* __restrict__ host_dst, const Rgb24* __restrict__ host_src, uint32_t w, uint32_t h) {

    assert(sizeof(Rgb24) == sizeof(uchar3));

    // TODO: Spare some memory!

    uchar3* dev_src = NULL;
    uchar3* dev_dst = NULL;
    float* dev_hue = NULL;
    float* dev_sat = NULL;
    float* dev_src_val = NULL;
    float* dev_dst_val = NULL;

    size_t dev_src_pitch = 0;
    size_t dev_dst_pitch = 0;
    size_t dev_hue_pitch = 0;
    size_t dev_sat_pitch = 0;
    size_t dev_src_val_pitch = 0;
    size_t dev_dst_val_pitch = 0;

    hipMallocPitch(&dev_src, &dev_src_pitch, w * sizeof dev_src[0], h);
    hipMallocPitch(&dev_dst, &dev_dst_pitch, w * sizeof dev_dst[0], h);
    hipMallocPitch(&dev_hue, &dev_hue_pitch, w * sizeof dev_hue[0], h);
    hipMallocPitch(&dev_sat, &dev_sat_pitch, w * sizeof dev_sat[0], h);
    hipMallocPitch(&dev_src_val, &dev_src_val_pitch, w * sizeof dev_src_val[0], h);
    hipMallocPitch(&dev_dst_val, &dev_dst_val_pitch, w * sizeof dev_dst_val[0], h);

    dev_src_tex.normalized = false;
    dev_src_tex.filterMode = hipFilterModePoint;
    dev_src_tex.addressMode[0] = hipAddressModeClamp;
    dev_src_tex.addressMode[1] = hipAddressModeClamp;
    dev_src_tex.addressMode[2] = hipAddressModeClamp;
    hipMemcpy2D(dev_src, dev_src_pitch, host_src, w * sizeof dev_src[0], w * sizeof dev_src[0], h, hipMemcpyHostToDevice);
    hipBindTexture2D(NULL, dev_src_tex, dev_src, w, h, dev_src_pitch);

    // 16*16 = 256 threads/tile
    // 32*32 = 1024 threads/tile
    const dim3 n_threads(32, 32);
    const dim3 n_blocks(
        (w + n_threads.x - 1) / n_threads.x,
        (h + n_threads.y - 1) / n_threads.y
    );

    // STEPS:
    // malloc-memset hist;
    //
    // per-pixel: RGB -> HSV
    // per-pixel: atomicInc(&hist[pixel])
    // inclusive scan: for l in 0..L: cdf[l] = ...
    // per-pixel: dst_val[i] = tone_map(src_val[i]);
    // per-pixel: HSV -> RGB

    // tex: src_rgb
    // buf: dev_hue
    // buf: dev_sat
    // buf: dev_src_val
    // buf: dev_dst_val

    // i: src_tex_rgb
    // o: dev_hue
    // o: dev_sat
    // o: dev_src_val
    rgb_to_hsv_gpu<<<n_blocks, n_threads>>>(
        dev_hue, dev_hue_pitch,
        dev_sat, dev_sat_pitch,
        dev_src_val, dev_src_val_pitch,
        w, h
    );
    // i: dev_src_val
    // o: dev_dst_val
    tone_map_gpu<<<n_blocks, n_threads>>>(
        dev_dst_val, dev_dst_val_pitch,
        dev_src_val, dev_src_val_pitch,
        w, h
    );
    // i: dev_hue
    // i: dev_sat
    // i: dev_dst_val
    // o: dev_dst_rgb
    hsv_to_rgb_gpu<<<n_blocks, n_threads>>>(
        dev_dst, dev_dst_pitch,
        dev_hue, dev_hue_pitch,
        dev_sat, dev_sat_pitch,
        dev_dst_val, dev_dst_val_pitch,
        w, h
    );

    hipMemcpy2D(host_dst, w * sizeof host_dst[0], dev_dst, dev_dst_pitch, w * sizeof dev_dst[0], h, hipMemcpyDeviceToHost);

    hipUnbindTexture(dev_src_tex);
    hipFree(dev_src);
    hipFree(dev_dst);
    hipFree(dev_hue);
    hipFree(dev_sat);
    hipFree(dev_src_val);
    hipFree(dev_dst_val);

    // TODO compare images (host_dst and dst_cpu)
}
#endif
