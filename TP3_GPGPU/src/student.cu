#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"

namespace IMAC
{
	// ==================================================== Ex 0
    __global__ void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax) {
        extern __shared__ uint sharedMem[];
        for(uint id = blockIdx.x * blockDim.x + threadIdx.x ; id < size ; id += gridDim.x * blockDim.x) {
            for(uint stride=1 ;  ; stride += stride) {
                const uint i = id * 2 * stride;
                const uint j = i + stride;
                if(i >= size || j >= size)
                    break;
                sharedMem[i] = max(sharedMem[i], sharedMem[j]);
                __syncthreads();
            }
        }
	}

	void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */)
    {
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(dev_array, array.size(), res1);
		
        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		
		std::cout << "========== Ex 2 " << std::endl;
		/// TODO

		std::cout << "========== Ex 3 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 4 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 5 " << std::endl;
		/// TODO
		

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}
